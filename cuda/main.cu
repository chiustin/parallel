#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <omp.h>
__device__ uint32_t rotate_left(uint32_t x, uint32_t n) {
    return  (x << n) | (x >> (32-n));
}
__device__ uint32_t encrypt(uint32_t m, uint32_t key) {
    return (rotate_left(m, key&31) + key)^key;
}
__host__ uint32_t h_rotate_left(uint32_t x, uint32_t n) {
    return  (x << n) | (x >> (32-n));
}
__host__ uint32_t h_encrypt(uint32_t m, uint32_t key) {
    return (h_rotate_left(m, key&31) + key)^key;
}
#define MAXN 16777216
#define GPULOCAL 128
#define BLOCKSZ (1024)
__global__ void vecdot(uint32_t keyA, uint32_t keyB, uint32_t C[], int N) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int l = x * BLOCKSZ;
    int r = l + BLOCKSZ;
    uint32_t sum = 0;
    if (r > N)	r = N;
    for (int i = l; i < r; i++)
        sum += encrypt(i, keyA) * encrypt(i, keyB);
    C[x] = sum;
}

uint32_t hostC[MAXN / GPULOCAL];
#define CheckErr(status) { gpuAssert((status), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, int abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
int main() {
    uint32_t N, keyA, keyB;
    uint32_t *cuArrC;
    hipMalloc((void **)&cuArrC, MAXN/GPULOCAL*sizeof(uint32_t));
    while (scanf("%u %u %u", &N, &keyA, &keyB) == 3) {
        int M = (N + BLOCKSZ-1) / BLOCKSZ;
        int LOCAL = GPULOCAL;
        M = (M + LOCAL) / LOCAL * LOCAL;
        dim3 cuBlock(LOCAL);
        dim3 cuGrid(M/LOCAL);
        vecdot<<<cuGrid, cuBlock>>>(keyA, keyB, cuArrC, N);
        CheckErr(hipGetLastError());
        hipMemcpy(hostC, cuArrC, M*sizeof(uint32_t), hipMemcpyDeviceToHost);
        uint32_t sum = 0;
#ifdef _OPENMP
        omp_set_num_threads(4);
#endif
        #pragma omp parallel for reduction(+: sum)
        for (int i = 0; i < M; i++)
            sum += hostC[i];
        printf("%u\n", sum);
    }
    hipFree(cuArrC);
    return 0;
}
